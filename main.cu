#include <stdio.h>
#include "sorting.cuh"
#include <stdlib.h>
#include <time.h>
#include <limits.h>
#include <assert.h>

#define S 64
#define K 64
#define L 64
#define CEIL(x,n) (x/n)*n + (n * (x % n > 0))
#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}

bool is_power_of_2(int x){
    return (x != 0) && ((x & (x - 1)) == 0);
}

int cmp(const void * a, const void * b){
    return *((int*)a) - *((int*)b);
}


int *get_splitters(int *in, int n){
    int *tmp = (int*) malloc(sizeof(int)*S*K);
    int *splitters = (int*) malloc(sizeof(int)*S);
    for(int i = 0; i < S*K; i++)
        tmp[i] = in[rand() % n];
    qsort(tmp,S*K,sizeof(int),cmp);
    for(int i=0; i < S; i++)
        splitters[i] = tmp[i*K];

    splitters[S-1] = INT_MAX;
    return splitters;
}



void warpSort(int *in, int n){
    assert(n % 64 == 0 && is_power_of_2(n));
    int sequence_size = 64;
    int size = sizeof(int) * n;
    int *out = (int*) malloc(size);

    int *splitters = get_splitters(in,n);
    int *d_in, *d_out;

    //step 1: bitonic sort
    hipMalloc(&d_in, size) ;
    hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
    bitonic_sort<<<n/sequence_size,sequence_size/4>>>(d_in,sequence_size);
    hipDeviceSynchronize();
    hipMemcpy(out, d_in, size, hipMemcpyDeviceToHost);

    //step 2: merge
    hipMalloc(&d_out, size) ;
    for(int seq = sequence_size; (n/seq) > L; seq *=2){
        merge<<<n/(seq*2),32>>>(d_in,d_out,seq);
        hipMemcpy(d_in, d_out, size, hipMemcpyDeviceToDevice);
    }
   
    hipDeviceSynchronize(); 
    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);



    //step 3: split into small tiles
    block_info **block_len = (block_info**) malloc(sizeof(block_info*) * S);
    for(int i = 0; i < S; i++)
        block_len[i] = (block_info*) calloc(L, sizeof(block_info));

    for(int i = 0; i<L; i++){
        int k = 0;
        for(int j = 0; j<S; j++){
            int start = k;
            while( k < (n/L) && out[(i*(n/L)) + k] <= splitters[j]) k++;
            block_len[j][i].start = (i*(n/L)) + start;
            block_len[j][i].end = (i*(n/L)) + k;
            block_len[j][i].len = k - start;
        }
    }

    int blocks_len[S] = {0};
    for(int s=0; s<S; s++){
        int sum = 0;
        for(int l=0; l<L; l++){
            sum += block_len[s][l].len;
        }
        blocks_len[s] = sum;
    }

    int *organized_input = (int *) calloc(n, sizeof(int));
    int z = 0;
    for(int j=0; j<S; j++){
        int start = 0;
        for(int i=0; i<L; i++){
            int a = z;
            for(int k=block_len[j][i].start; k < block_len[j][i].end; k++)
                organized_input[z++] = out[k];
            block_len[j][i].start = start;
            block_len[j][i].end = start + z - a;
            start = start + z - a;
        }
    }

    int **d_ins;
    int **d_outs;
    d_ins = (int**) malloc(sizeof(int*) * S);
    d_outs = (int**) malloc(sizeof(int*) * S);
    int offset = 0;
    for(int i =0; i<S; i++){
        hipMalloc(&d_ins[i],sizeof(int)*blocks_len[i]);
        hipMemcpy(d_ins[i],organized_input + offset,sizeof(int) * blocks_len[i],hipMemcpyHostToDevice);
        hipMalloc(&d_outs[i],sizeof(int)*blocks_len[i]);
        offset += blocks_len[i];
    }

    block_info **d_block_len = (block_info**) malloc(sizeof(block_info*) * S);
    hipStream_t stream[S];
    for(int i = 0; i < S; i++){
        hipStreamCreate(&stream[i]);
        hipMalloc(&d_block_len[i],sizeof(block_info) * L);
        hipMemcpy(d_block_len[i],block_len[i],sizeof(block_info) * L,hipMemcpyHostToDevice);
    }


    //step 4: merge independent S sequences
    offset = 0;
    for(int s = 0; s < S; s++){
        for(int k = L/2; k > 0; k /= 2){
            final_merge<<<k,32,0,stream[s]>>>(d_ins[s],d_outs[s],d_block_len[s],L);
            hipMemcpyAsync(d_ins[s],d_outs[s],blocks_len[s]*sizeof(int),hipMemcpyDeviceToDevice,stream[s]);
        }
        hipMemcpyAsync(out + offset,d_outs[s],sizeof(int) * blocks_len[s],hipMemcpyDeviceToHost,stream[s]);
        offset += blocks_len[s];
    }
    hipDeviceSynchronize();  
    memcpy(in,out,sizeof(int)*n);
}

int main(){
    srand(time(NULL));
    int sequence_size = 128;
    int n = sequence_size * (1 << 16);
    int size = sizeof(int) * n;
    int *in = (int*) malloc(size);
    for(int i=0; i<n; i++) in[i] = rand() % 10000;
    int *in2 = (int*) malloc(size);
    printf("testing on %d elements\n",n);
    memcpy(in2,in,sizeof(int)*n); 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    warpSort(in,n);
    cudaCheckError();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("warpsort time ms: %f\n",milliseconds);

    
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start); 
    qsort(in2,n,sizeof(int),cmp);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("sequential quicksort time ms: %f\n",milliseconds);

    for(int i=0; i<n;i++)
        assert(in[i] == in2[i]);

    return 0;
}
