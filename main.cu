#include <stdio.h>
#include "sorting.cuh"
#include <stdlib.h>
#include <time.h>
#include <limits.h>
#include <assert.h>

#define S 64
#define K 64
#define L 64
#define CEIL(x,n) (x/n)*n + (n * (x % n > 0))

int cmp(const void * a, const void * b){
    return *((int*)a) - *((int*)b);
}

typedef struct{
    int len;
    int start;
    int end;
}block_info;

void test_arb_merge(){
    for(int y = 0; y < 100; y++){
        int arr_size = (rand() % 2046);
        int *d_in, *d_out;
        int *myvect= (int*) malloc(sizeof(int)*arr_size);
        int a_size = (rand() % arr_size) + 1;
        printf("arr_size: %d, a_size: %d\n",arr_size, a_size);

        for(int i=0;i<a_size;i++) myvect[i] = i;
        for(int i=a_size;i<arr_size;i++) myvect[i] = i - a_size;
        int *tocmp = (int*) malloc(sizeof(int)*arr_size);
        memcpy(tocmp,myvect,sizeof(int)*arr_size);
        
        hipMalloc(&d_out, sizeof(int) * arr_size);
        hipMalloc(&d_in, sizeof(int) * arr_size);
        hipMemcpy(d_in,myvect,sizeof(int)*arr_size,hipMemcpyHostToDevice);
        arb_merge<<<1,32>>>(d_in,d_out,0,a_size,0,arr_size - a_size,a_size);
        hipDeviceSynchronize();
        int *o = (int*) malloc(sizeof(int)*arr_size);
        hipMemcpy(o,d_out,sizeof(int)*arr_size,hipMemcpyDeviceToHost);
        
        qsort(tocmp,arr_size, sizeof(int),cmp); 
        if(memcmp(tocmp,o,sizeof(int)*arr_size) != 0)
            printf("diversi\n");
    }
}

int *get_splitters(int *in, int n){
    int *tmp = (int*) malloc(sizeof(int)*S*K);
    int *splitters = (int*) malloc(sizeof(int)*S);
    for(int i = 0; i < S*K; i++)
        tmp[i] = in[rand() % n];
    qsort(tmp,S*K,sizeof(int),cmp);
    for(int i=0; i < S; i++)
        splitters[i] = tmp[i*K];

    splitters[S-1] = INT_MAX;
    return splitters;
}

int main(){
    srand(time(NULL));
    int sequence_size = 64;
    int n = sequence_size * 128;
    int size = sizeof(int) * n;
    int *in = (int*) malloc(size);
    int *out = (int*) malloc(size);
    for(int i=0; i<n; i++) in[i] = rand() % 1000;
    int *splitters = get_splitters(in,n);
    int *d_in, *d_out;

    hipMalloc(&d_in, size) ;
    hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
    bitonic_sort<<<n/sequence_size,sequence_size/4>>>(d_in,sequence_size);
    hipDeviceSynchronize();
    hipMemcpy(out, d_in, size, hipMemcpyDeviceToHost);

    for(int i=0; i<(n/sequence_size); i++){
        for(int j=0; j < sequence_size - 1; j++)
            assert(out[(i*sequence_size) + j] <= out[(i*sequence_size) +j +1]);
    }

    hipMalloc(&d_out, size) ;
    for(int seq = sequence_size; (n/seq) > L; seq *=2){
        merge<<<n/(seq*2),32>>>(d_in,d_out,seq);
        hipMemcpy(d_in, d_out, size, hipMemcpyDeviceToDevice);
    }
   
    hipDeviceSynchronize(); 
    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);


    int *splitters_indexes = (int*) malloc(sizeof(int) * S);

    printf("splitters: ");
    for(int i=0; i  < S; i++){
        printf("%d ",splitters[i]);
    }
    printf("\n");

    // INIZIALIZZAZIONE BLOCCHI
    block_info **block_len = (block_info**) malloc(sizeof(block_info*) * L);
    for(int i = 0; i < L; i++)
        block_len[i] = (block_info*) calloc(S, sizeof(block_info));

    for(int i = 0; i<L; i++){
        int k = 0;
        for(int j = 0; j<S; j++){
            int start = k;
            while( k < (n/L) && out[(i*(n/L)) + k] <= splitters[j]) k++;
            block_len[i][j].start = (i*(n/L)) + start;
            block_len[i][j].end = (i*(n/L)) + k;
            block_len[i][j].len = k - start;
        }
    }
    
    for(int i=0; i<L; i++){
        int sum = 0;
        for(int j=0; j<S; j++)
            sum += block_len[i][j].len;
        assert(sum == (n/L));
    }

    int blocks_len[S] = {0};
    for(int s=0; s<S; s++){
        int sum = 0;
        for(int l=0; l<L; l++){
            sum += block_len[l][s].len;
        }
        blocks_len[s] = sum;
    }

    // FINE INIZIALIZZAZIONE BLOCCHI

    hipMemset( d_out, 0,sizeof(int)*n);

    int *organized_input = (int *) calloc(n, sizeof(int));
    int z = 0;
    for(int j=0; j<S; j++){
        int start = 0;
        for(int i=0; i<L; i++){
            int a = z;
            for(int k=block_len[i][j].start; k < block_len[i][j].end; k++)
                organized_input[z++] = out[k];
            block_len[i][j].start = start;
            block_len[i][j].end = start + z - a;
            start = start + z - a;
        }
    }
    //return 0;

    int **d_ins;
    int **d_outs;
    d_ins = (int**) malloc(sizeof(int*) * S);
    d_outs = (int**) malloc(sizeof(int*) * S);
    int offset = 0;
    for(int i =0; i<S; i++){
        hipMalloc(&d_ins[i],sizeof(int)*blocks_len[i]);
        hipMemcpy(d_ins[i],organized_input + offset,sizeof(int) * blocks_len[i],hipMemcpyHostToDevice);
        hipMalloc(&d_outs[i],sizeof(int)*blocks_len[i]);
        offset += blocks_len[i];
    }

//    organized_input = organized_input + blocks_len[0];
//    hipMemcpy(d_in, organized_input, size, hipMemcpyHostToDevice);
    for(int s = 0; s < S; s++){
        for(int k=1; k <= (L/2); k*=2){
            for(int i = 0; i< L-k; i+=(k*2)){
                arb_merge<<<1,32>>>(d_ins[s],d_outs[s],block_len[i][s].start,block_len[i][s].len, block_len[i][s].start, block_len[i+k][s].len,block_len[i+k][s].start);
                hipDeviceSynchronize();
                hipMemcpy(out, d_outs[1], blocks_len[1]*sizeof(int), hipMemcpyDeviceToHost);
                block_len[i][s].end = block_len[i+k][s].end;
                block_len[i][s].len = block_len[i][s].len + block_len[i+k][s].len;
                memset(&block_len[i+k][s],0,sizeof(block_info));
            }
            hipMemcpy(d_ins[s],d_outs[s],blocks_len[s]*sizeof(int),hipMemcpyDeviceToDevice);
            hipMemcpy(organized_input,d_outs[s],blocks_len[s]*sizeof(int),hipMemcpyDeviceToHost);
        }
    }

    hipDeviceSynchronize(); 
    
  //  hipMemcpy(out, d_outs[1], blocks_len[1]*sizeof(int), hipMemcpyDeviceToHost);
    offset = 0;
    for(int i =0; i<S; i++){
        hipMemcpy(out + offset,d_outs[i],sizeof(int) * blocks_len[i],hipMemcpyDeviceToHost);
        offset += blocks_len[i];
    }
    for(int i=0; i<n;i++)
        printf("%d ",out[i]);

    return 0;
}
