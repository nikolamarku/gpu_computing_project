#include "hip/hip_runtime.h"
#include <stdio.h>
#include "sorting.cuh"
#include <stdlib.h>
#include <time.h>
#include <limits.h>
#include <assert.h>
#include <time.h>

#define SEQUENCE_SIZE 128
#define S 128
#define K 64
#define L 64
#define CEIL(x,n) (x/n)*n + (n * (x % n > 0))
#define CHECK_PTR(ptr) {\
    if(ptr == NULL){ \
        printf("ptr is null %s:%d\n",__FILE__,__LINE__); \
        exit(1); \
    } \
}

#define CHECK(x) { \
    x;                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}

bool is_power_of_2(int x){
    return (x != 0) && ((x & (x - 1)) == 0);
}

int cmp(const void * a, const void * b){
    return *((int*)a) - *((int*)b);
}


int *get_splitters(int *in, int n){
    int *tmp = (int*) malloc(sizeof(int)*S*K);
    int *splitters = (int*) malloc(sizeof(int)*S);
    for(int i = 0; i < S*K; i++)
        tmp[i] = in[rand() % n];
    qsort(tmp,S*K,sizeof(int),cmp);
    for(int i=0; i < S; i++)
        splitters[i] = tmp[i*K];

    splitters[S-1] = INT_MAX;
    return splitters;
}



void warpSort(int *in, int n){
    assert(n % 64 == 0 && is_power_of_2(n));
    int size = sizeof(int) * n;
    int *out = (int*) malloc(size);

    int *splitters = get_splitters(in,n);
    int *d_in, *d_out;

    //step 1: bitonic sort
    CHECK(hipMalloc(&d_in, size));
    CHECK(hipMemcpy(d_in, in, size, hipMemcpyHostToDevice));
    bitonic_sort<<<n/SEQUENCE_SIZE,SEQUENCE_SIZE/4>>>(d_in,SEQUENCE_SIZE);
    CHECK(hipMemcpy(out, d_in, size, hipMemcpyDeviceToHost));

    //step 2: merge
    CHECK(hipMalloc(&d_out, size));
    for(int seq = SEQUENCE_SIZE; (n/seq) > L; seq *=2){
        merge<<<n/(seq*2),32>>>(d_in,d_out,seq);
        CHECK(hipMemcpy(d_in, d_out, size, hipMemcpyDeviceToDevice));
    }
    CHECK(hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost));
    CHECK(hipFree(d_out));


    //step 3: split into small tiles
    block_info **block_len = (block_info**) malloc(sizeof(block_info*) * S);
    int blocks_len[S] = {0};
    for(int i = 0; i < S; i++)
        block_len[i] = (block_info*) calloc(L, sizeof(block_info));

    for(int i = 0; i<L; i++){
        int k = 0;
        for(int j = 0; j<S; j++){
            int start = k;
            while( k < (n/L) && out[(i*(n/L)) + k] <= splitters[j]) k++;
            block_len[j][i].start = (i*(n/L)) + start;
            block_len[j][i].end = (i*(n/L)) + k;
            block_len[j][i].len = k - start;
            blocks_len[j] += (k -start);
        }
    }
    //re arrange
    int *organized_input = (int *) calloc(n, sizeof(int));
    int z = 0;
    for(int j=0; j<S; j++){
        int start = 0;
        for(int i=0; i<L; i++){
            int a = z;
            for(int k=block_len[j][i].start; k < block_len[j][i].end; k++)
                organized_input[z++] = out[k];
            block_len[j][i].start = start;
            block_len[j][i].end = start + z - a;
            start = start + z - a;
        }
    }

    int **h_ins;
    int **h_outs;
    h_ins = (int**) malloc(sizeof(int*) * S);
    h_outs = (int**) malloc(sizeof(int*) * S);
    int offset = 0;
    for(int i =0; i<S; i++){
        CHECK(hipMalloc(&h_ins[i],sizeof(int)*blocks_len[i]));
        CHECK(hipMemcpy(h_ins[i],organized_input + offset,sizeof(int) * blocks_len[i],hipMemcpyHostToDevice));
        CHECK(hipMalloc(&h_outs[i],sizeof(int)*blocks_len[i]));
        offset += blocks_len[i];
    }

    block_info **h_block_len = (block_info**) malloc(sizeof(block_info*) * S);
    for(int i = 0; i < S; i++){
        CHECK(hipMalloc(&h_block_len[i],sizeof(block_info) * L));
        CHECK(hipMemcpy(h_block_len[i],block_len[i],sizeof(block_info) * L,hipMemcpyHostToDevice));
    }


    //step 4: merge independent S sequences
    int **d_ins;
    int **d_outs;
    block_info **dd_block_info;
    hipMalloc(&d_ins, sizeof(int*)*S);
    hipMalloc(&d_outs, sizeof(int*)*S);
    hipMalloc(&dd_block_info, sizeof(block_info*)*S);
    
    hipMemcpy(d_ins, h_ins, sizeof(int*)*S,hipMemcpyHostToDevice);
    hipMemcpy(d_outs, h_outs, sizeof(int*)*S, hipMemcpyHostToDevice);
    hipMemcpy(dd_block_info, h_block_len, sizeof(int*)*S, hipMemcpyHostToDevice);

    for(int k = L/2; k > 0; k /= 2){
        dim3 grid(S,k);
        final_merge<<<grid,32>>>(d_ins,d_outs,dd_block_info,L);
        CHECK();
        for(int i = 0; i < S; i++)
            CHECK(hipMemcpy(h_ins[i],h_outs[i],blocks_len[i]*sizeof(int),hipMemcpyDeviceToDevice));
    }

    offset = 0;
    for(int s = 0; s < S; s++){
        CHECK(hipMemcpy(out + offset,h_outs[s],sizeof(int) * blocks_len[s],hipMemcpyDeviceToHost));
        offset += blocks_len[s];
    }
    hipDeviceSynchronize();
    memcpy(in,out,sizeof(int)*n);
}

int main(int argc, char **argv){
    srand(time(NULL));
    int n = SEQUENCE_SIZE * (1 << atoi(argv[1]));
    int size = sizeof(int) * n;
    int *in = (int*) malloc(size);
    CHECK_PTR(in);
    for(int i=0; i<n; i++) in[i] = rand() % 10000;
    int *in2 = (int*) malloc(size);
    CHECK_PTR(in2);
    memcpy(in2,in,sizeof(int)*n); 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    warpSort(in,n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float warpsort_milliseconds = 0;
    hipEventElapsedTime(&warpsort_milliseconds, start, stop);

    clock_t begin = clock();
    qsort(in2,n,sizeof(int),cmp);
    clock_t end = clock();
    double qsort_seconds = (double)(end - begin) / CLOCKS_PER_SEC;

    assert(memcmp(in,in2,sizeof(int)*n) == 0);
    printf("%d;%f;%f\n",n,warpsort_milliseconds/1000,qsort_seconds);
    return 0;
}
