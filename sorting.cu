#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdbool.h>
#include <assert.h>
#include "sorting.cuh"


__device__ void phase(int *x, int sequence_size, int comparator_size, bool two_ways, bool full){
    int MAX_COMPARATOR_N = (sequence_size/2)/2;
    int idx = threadIdx.x;
    int num_of_comparators = (sequence_size / comparator_size) /2;
    //fix this
    int selected_comparator = idx / (MAX_COMPARATOR_N/num_of_comparators);

    for (int j = comparator_size / 2; j > 0; j /= 2) {
        int groups_in_comp_stage = comparator_size / (j*2);
        int selected_group = idx % groups_in_comp_stage;
        int group_start = selected_group * (2*j);
        int offset_in_group = (idx/groups_in_comp_stage) % j;
        int k0 = (selected_comparator * (comparator_size*2)) + group_start + offset_in_group;
        int k1 = sequence_size - 1 - k0;
        int a = x[k0], b = x[k0+j];
        x[k0] =   min(a,b);
        x[k0+j] = max(a,b);
        if(full){
          int c= x[k1-j], d = x[k1];
          x[k1-j] = (1-two_ways)*min(c,d) + two_ways*max(c,d);
          x[k1] =   (1-two_ways)*max(c,d) + two_ways*min(c,d);
        }
    }
}

__global__ void bitonic_sort(int *x, int sequence_size){
    assert(blockDim.x == (sequence_size/4));
    int offset = blockIdx.x * sequence_size;
    for (int i = 2; i < sequence_size; i *= 2)
        phase(x + offset,sequence_size,i,true,true);
    phase(x + offset,sequence_size,sequence_size,false,true);
}

__global__ void merge(int *x, int *out, int sequence_size){
    assert(blockDim.x == 32);
    assert(sequence_size % 32 == 0 && sequence_size >= 64);
    int offset = blockIdx.x * (sequence_size*2);
    int idx = threadIdx.x;
    int *vect = x + offset;
    out = out + offset;
    int *A = vect, *B = vect + sequence_size;
    __shared__ int tile[64];
    tile[32 - 1 - idx] =  A[idx];
	tile[32 + idx]     =  B[idx];
    int max_A = tile[0], max_B = tile[63];
    int A_cursor = 32, B_cursor = 32;

    while(A_cursor < sequence_size || B_cursor < sequence_size ){
      phase(tile,64,64,false,false);
      out[ A_cursor + B_cursor - 64 + idx] = tile[idx];
      if((max_A <= max_B && A_cursor < sequence_size) || B_cursor == sequence_size){
        assert(A_cursor < sequence_size);
        tile[32 - 1 - idx] =  A[idx + A_cursor];
        max_A = tile[0];
        A_cursor += 32;
      }else{
        assert(B_cursor < sequence_size);
        tile[32 - 1 - idx] = B[idx + B_cursor];
        max_B = tile[0];
        B_cursor += 32;
      }
    }

    phase(tile,64,64,false,false);
    out[A_cursor + B_cursor - 64 + idx] = tile[idx];
    out[A_cursor + B_cursor - 64 + idx + 32 ] = tile[idx + 32];
}



__global__ void final_merge(int *x, int *out, block_info* b_info, const int L){
    assert(blockDim.x == 32);
    int i = blockIdx.x * (L/gridDim.x);
    int k = (L/2) / gridDim.x;
    int idx = threadIdx.x;

    int offset = b_info[i].start,
        seq_a_size = b_info[i].len,
        seq_a_pos = b_info[i].start,
        seq_b_size = b_info[i+k].len, 
        seq_b_pos = b_info[i+k].start;
    out = out + offset;

    int *A = x + seq_a_pos, *B = x + seq_b_pos;
    __shared__ int tile[64];
    tile[32 - 1 - idx] =  (idx < seq_a_size) * A[idx] + (1-(idx < seq_a_size)) * INT_MAX;
	tile[32 + idx]     =  (idx < seq_b_size) * B[idx] + (1-(idx < seq_b_size)) *  INT_MAX;
    int max_A = tile[0], max_B = tile[63];
    int A_cursor = min(seq_a_size,32);
    int B_cursor = min(seq_b_size, 32);
    int copied = 0;

    while(A_cursor < seq_a_size || B_cursor < seq_b_size ){
        phase(tile,64,64,false,false);
        out[ copied + idx ] = tile[idx];
        copied+=32;
        if((max_A <= max_B && A_cursor < seq_a_size) || B_cursor == seq_b_size){
            assert(A_cursor < seq_a_size);
            tile[32 - 1 - idx] =  ((idx+A_cursor) < seq_a_size) ? A[idx + A_cursor] : INT_MAX;
            max_A = tile[0];
            A_cursor = min(A_cursor + 32, seq_a_size);
        }else{
            assert(B_cursor < seq_b_size);
            tile[32 - 1 - idx] = ((idx+B_cursor) < seq_b_size) ? B[idx + B_cursor] : INT_MAX;
            max_B = tile[0];
            B_cursor = min(B_cursor + 32, seq_b_size);
        }
    }

    phase(tile,64,64,false,false);

    //check if it fits output
    if(tile[idx] < INT_MAX)
        out[copied + idx] = tile[idx];
    if(tile[idx+32] < INT_MAX)
        out[copied + idx + 32] = tile[idx + 32];


    b_info[i].end = b_info[i+k].end;
    b_info[i].len = b_info[i].len + b_info[i+k].len;
}